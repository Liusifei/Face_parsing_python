#include "hip/hip_runtime.h"
/*
 * Author: Liangji 
 * Email: liangji20040249@gmail.com
*/
#include <vector>

#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/gaterecurrent2d_layer.hpp"
#include "caffe/util/io.hpp"

namespace caffe {



__device__ void get_gate_idx(int h1,int w1,int h2,int w2, int * out,bool horizontal, bool reverse)
{
	if(horizontal && ! reverse) // left -> right
	{
		if(w1>w2)
		{
			out[0]=h1;
			out[1]=w1;
		}
		else
		{
			out[0]=h2;
			out[1]=w2;
		}
	}
	if(horizontal && reverse)  // right -> left
	{
		if(w1<w2)
		{
			out[0]=h1;
			out[1]=w1;
		}
		else
		{
			out[0]=h2;
			out[1]=w2;
		}
	}
	if(!horizontal && !reverse)  // top  -> bottom
	{
		if(h1>h2)
		{
			out[0]=h1;
			out[1]=w1;
		}
		else
		{
			out[0]=h2;
			out[1]=w2;
		}
	}
	if(!horizontal && reverse)  // bottom -> top
	{
		if(h1<h2)
		{
			out[0]=h1;
			out[1]=w1;
		}
		else
		{
			out[0]=h2;
			out[1]=w2;
		}
	}

}

template <typename Dtype>
__device__ Dtype get_data(Dtype * data, int num, int channels,int height, int width,int n,int c,int h,int w)
{
	if(h<0 || h >=height)
		return 0;
	if(w<0 || w >= width)
		return 0;
	
	return data[n*channels*height*width + c * height*width + h * width + w];
}

template <typename Dtype>
__device__ void set_data(Dtype * data, int num, int channels,int height, int width,int n,int c,int h,int w,Dtype v)
{
	if(h<0 || h >=height)
		return ;
	if(w<0 || w >= width)
		return ;
	
	data[n*channels*height*width + c * height*width + h * width + w]=v;
}

template <typename Dtype>
__device__ Dtype get_gate(Dtype * data, int num, int channels,int height, int width,int n,int c,int h1,int w1,int h2,int w2,bool horizontal,bool reverse)
{
	if(h1<0 || h1 >=height)
		return 0;
	if(w1<0 || w1 >= width)
		return 0;
	if(h2<0 || h2 >=height)
		return 0;
	if(w2<0 || w2 >= width)
		return 0;
	int idx[2];
		
	get_gate_idx(h1,w1,h2,w2, idx,horizontal, reverse);

	int h = idx[0];
	int w = idx[1];
	
	return data[n*channels*height*width + c * height*width + h * width + w];
}
template <typename Dtype>
__device__ void set_gate(Dtype * data, int num, int channels,int height, int width,int n,int c,int h1,int w1,int h2,int w2,bool horizontal,bool reverse,Dtype v)
{
	if(h1<0 || h1 >=height)
		return ;
	if(w1<0 || w1 >= width)
		return ;
	if(h2<0 || h2 >=height)
		return ;
	if(w2<0 || w2 >= width)
		return ;
	int idx[2];
		
	get_gate_idx(h1,w1,h2,w2, idx,horizontal, reverse);

	int h = idx[0];
	int w = idx[1];
	
	data[n*channels*height*width + c * height*width + h * width + w]=v;
}
template <typename Dtype>
__device__ void set_gate_add(Dtype * data, int num, int channels,int height, int width,int n,int c,int h1,int w1,int h2,int w2,bool horizontal,bool reverse,Dtype v)
{
	if(h1<0 || h1 >=height)
		return ;
	if(w1<0 || w1 >= width)
		return ;
	if(h2<0 || h2 >=height)
		return ;
	if(w2<0 || w2 >= width)
		return ;
	int idx[2];
		
	get_gate_idx(h1,w1,h2,w2, idx,horizontal, reverse);

	int h = idx[0];
	int w = idx[1];
	
	atomicAdd((float *)(data + n*channels*height*width + c * height*width + h * width + w),float(v));
}


template <typename Dtype>
__global__ void forward_one_col_left_right(const int count, int T, int num,int channels, int height,  int width,const Dtype* X,const Dtype* G1, const Dtype* G2,const Dtype* G3,const Dtype* Idx, Dtype* H,bool horizontal,bool reverse) {
CUDA_KERNEL_LOOP(index, count) {

	
	int hc_count = height * channels;

	int n,c,h,w;
	int temp=index;
	w = T;
	n = temp / hc_count;
	temp = temp % hc_count;
	c = temp / height;
	temp = temp % height;
	h = temp;
	

	Dtype x_data = get_data(X,num,channels,height,width,n,c,h,w);

	
	
	Dtype g1_idx = get_gate(Idx,num,3,height,width,n,0,h,w,h-1,w-1,horizontal,reverse);
	Dtype g2_idx = get_gate(Idx,num,3,height,width,n,1,h,w,h,w-1,horizontal,reverse);
	Dtype g3_idx = get_gate(Idx,num,3,height,width,n,2,h,w,h+1,w-1,horizontal,reverse);

	
	Dtype g_data_1 = get_gate(G1,num,channels,height,width,n,c,h,w,h-1,w-1,horizontal,reverse);
	Dtype h_minus1_data_1 = get_data(H,num,channels,height,width,n,c,h-1,w-1);
	Dtype h1 = (1-g_data_1)*x_data + g_data_1 * h_minus1_data_1;

	Dtype g_data_2 = get_gate(G2,num,channels,height,width,n,c,h,w,h,w-1,horizontal,reverse);
	Dtype h_minus1_data_2 = get_data(H,num,channels,height,width,n,c,h,w-1);
	Dtype h2 = (1-g_data_2)*x_data + g_data_2 * h_minus1_data_2;

	Dtype g_data_3 = get_gate(G3,num,channels,height,width,n,c,h,w,h+1,w-1,horizontal,reverse);
	Dtype h_minus1_data_3 = get_data(H,num,channels,height,width,n,c,h+1,w-1);
	Dtype h3 = (1-g_data_3)*x_data + g_data_3 * h_minus1_data_3;

	Dtype h_data = h1*g1_idx + h2 * g2_idx + h3*g3_idx;

	set_data(H,num,channels,height,width,n,c,h,w,h_data);
	

}
}

template <typename Dtype>
__global__ void forward_one_col_right_left(const int count, int T, int num,int channels, int height,  int width,const Dtype* X,const Dtype* G1, const Dtype* G2,const Dtype* G3,const Dtype* Idx, Dtype* H,bool horizontal,bool reverse) {
CUDA_KERNEL_LOOP(index, count) {

	
	int hc_count = height * channels;

	int n,c,h,w;
	int temp=index;
	w = T;
	n = temp / hc_count;
	temp = temp % hc_count;
	c = temp / height;
	temp = temp % height;
	h = temp;
	

	Dtype x_data = get_data(X,num,channels,height,width,n,c,h,w);

	
	
	Dtype g1_idx = get_gate(Idx,num,3,height,width,n,0,h,w,h-1,w+1,horizontal,reverse);
	Dtype g2_idx = get_gate(Idx,num,3,height,width,n,1,h,w,h,w+1,horizontal,reverse);
	Dtype g3_idx = get_gate(Idx,num,3,height,width,n,2,h,w,h+1,w+1,horizontal,reverse);

	
	Dtype g_data_1 = get_gate(G1,num,channels,height,width,n,c,h,w,h-1,w+1,horizontal,reverse);
	Dtype h_minus1_data_1 = get_data(H,num,channels,height,width,n,c,h-1,w+1);
	Dtype h1 = (1-g_data_1)*x_data + g_data_1 * h_minus1_data_1;

	Dtype g_data_2 = get_gate(G2,num,channels,height,width,n,c,h,w,h,w+1,horizontal,reverse);
	Dtype h_minus1_data_2 = get_data(H,num,channels,height,width,n,c,h,w+1);
	Dtype h2 = (1-g_data_2)*x_data + g_data_2 * h_minus1_data_2;

	Dtype g_data_3 = get_gate(G3,num,channels,height,width,n,c,h,w,h+1,w+1,horizontal,reverse);
	Dtype h_minus1_data_3 = get_data(H,num,channels,height,width,n,c,h+1,w+1);
	Dtype h3 = (1-g_data_3)*x_data + g_data_3 * h_minus1_data_3;

	Dtype h_data = h1*g1_idx + h2 * g2_idx + h3*g3_idx;

	set_data(H,num,channels,height,width,n,c,h,w,h_data);
	

}
}

template <typename Dtype>
__global__ void forward_one_row_top_bottom(const int count, int T, int num,int channels, int height,  int width,const Dtype* X,const Dtype* G1, const Dtype* G2,const Dtype* G3,const Dtype* Idx, Dtype* H,bool horizontal,bool reverse) {
CUDA_KERNEL_LOOP(index, count) {

	
	int wc_count = width * channels;

	int n,c,h,w;
	int temp=index;
	h = T;
	n = temp / wc_count;
	temp = temp % wc_count;
	c = temp / width;
	temp = temp % width;
	w = temp;
	

	Dtype x_data = get_data(X,num,channels,height,width,n,c,h,w);

	
	
	Dtype g1_idx = get_gate(Idx,num,3,height,width,n,0,h,w,h-1,w-1,horizontal,reverse);
	Dtype g2_idx = get_gate(Idx,num,3,height,width,n,1,h,w,h-1,w,horizontal,reverse);
	Dtype g3_idx = get_gate(Idx,num,3,height,width,n,2,h,w,h-1,w+1,horizontal,reverse);

	
	Dtype g_data_1 = get_gate(G1,num,channels,height,width,n,c,h,w,h-1,w-1,horizontal,reverse);
	Dtype h_minus1_data_1 = get_data(H,num,channels,height,width,n,c,h-1,w-1);
	Dtype h1 = (1-g_data_1)*x_data + g_data_1 * h_minus1_data_1;

	Dtype g_data_2 = get_gate(G2,num,channels,height,width,n,c,h,w,h-1,w,horizontal,reverse);
	Dtype h_minus1_data_2 = get_data(H,num,channels,height,width,n,c,h-1,w);
	Dtype h2 = (1-g_data_2)*x_data + g_data_2 * h_minus1_data_2;

	Dtype g_data_3 = get_gate(G3,num,channels,height,width,n,c,h,w,h-1,w+1,horizontal,reverse);
	Dtype h_minus1_data_3 = get_data(H,num,channels,height,width,n,c,h-1,w+1);
	Dtype h3 = (1-g_data_3)*x_data + g_data_3 * h_minus1_data_3;

	Dtype h_data = h1*g1_idx + h2 * g2_idx + h3*g3_idx;

	set_data(H,num,channels,height,width,n,c,h,w,h_data);
	

}
}



template <typename Dtype>
__global__ void forward_one_row_bottom_top(const int count, int T, int num,int channels, int height,  int width,const Dtype* X,const Dtype* G1, const Dtype* G2,const Dtype* G3,const Dtype* Idx, Dtype* H,bool horizontal,bool reverse) {
CUDA_KERNEL_LOOP(index, count) {

	
	int wc_count = width * channels;

	int n,c,h,w;
	int temp=index;
	h = T;
	n = temp / wc_count;
	temp = temp % wc_count;
	c = temp / width;
	temp = temp % width;
	w = temp;
	

	Dtype x_data = get_data(X,num,channels,height,width,n,c,h,w);

	
	
	Dtype g1_idx = get_gate(Idx,num,3,height,width,n,0,h,w,h+1,w-1,horizontal,reverse);
	Dtype g2_idx = get_gate(Idx,num,3,height,width,n,1,h,w,h+1,w,horizontal,reverse);
	Dtype g3_idx = get_gate(Idx,num,3,height,width,n,2,h,w,h+1,w+1,horizontal,reverse);

	
	Dtype g_data_1 = get_gate(G1,num,channels,height,width,n,c,h,w,h+1,w-1,horizontal,reverse);
	Dtype h_minus1_data_1 = get_data(H,num,channels,height,width,n,c,h+1,w-1);
	Dtype h1 = (1-g_data_1)*x_data + g_data_1 * h_minus1_data_1;

	Dtype g_data_2 = get_gate(G2,num,channels,height,width,n,c,h,w,h+1,w,horizontal,reverse);
	Dtype h_minus1_data_2 = get_data(H,num,channels,height,width,n,c,h+1,w);
	Dtype h2 = (1-g_data_2)*x_data + g_data_2 * h_minus1_data_2;

	Dtype g_data_3 = get_gate(G3,num,channels,height,width,n,c,h,w,h+1,w+1,horizontal,reverse);
	Dtype h_minus1_data_3 = get_data(H,num,channels,height,width,n,c,h+1,w+1);
	Dtype h3 = (1-g_data_3)*x_data + g_data_3 * h_minus1_data_3;

	Dtype h_data = h1*g1_idx + h2 * g2_idx + h3*g3_idx;

	set_data(H,num,channels,height,width,n,c,h,w,h_data);
	

}
}


template <typename Dtype>
__global__ void backward_one_col_left_right(const int count, int T, int num,int channels, int height,  int width,const Dtype* X,const Dtype* G1, const Dtype* G2,const Dtype* G3,const Dtype* Idx, const Dtype* H, Dtype * X_diff, Dtype * G1_diff,Dtype* G2_diff,Dtype * G3_diff,Dtype * Idx_diff,  Dtype * Hdiff,bool horizontal,bool reverse) {
CUDA_KERNEL_LOOP(index, count) {

	
	int hc_count = height * channels;

	int n,c,h,w;
	int temp=index;



	w = T;
	n = temp / hc_count;
	temp = temp % hc_count;
	c = temp / height;
	temp = temp % height;
	h = temp;


	Dtype x_data = get_data(X,num,channels,height,width,n,c,h,w);

	//h(t)_diff = top(t)_diff
	Dtype h_diff = get_data(Hdiff,num,channels,height,width,n,c,h,w); 

	//h(t)_diff += h(t+1)_diff * g(t+1) if t<T
	Dtype add1_g3_idx = get_gate(Idx,num,3,height,width,n,2,h,w,h-1,w+1,horizontal,reverse);
	Dtype add1_h3_diff = get_data(Hdiff,num,channels,height,width,n,c,h-1,w+1);
	Dtype add1_g3_data = get_gate(G3,num,channels,height,width,n,c,h,w,h-1,w+1,horizontal,reverse);

	Dtype add1_g2_idx = get_gate(Idx,num,3,height,width,n,1,h,w,h,w+1,horizontal,reverse);
	Dtype add1_h2_diff = get_data(Hdiff,num,channels,height,width,n,c,h,w+1);
	Dtype add1_g2_data = get_gate(G2,num,channels,height,width,n,c,h,w,h,w+1,horizontal,reverse);

	Dtype add1_g1_idx = get_gate(Idx,num,3,height,width,n,0,h,w,h+1,w+1,horizontal,reverse);
	Dtype add1_h1_diff = get_data(Hdiff,num,channels,height,width,n,c,h+1,w+1);
	Dtype add1_g1_data = get_gate(G1,num,channels,height,width,n,c,h,w,h+1,w+1,horizontal,reverse);

	h_diff = h_diff + add1_g3_idx * add1_h3_diff * add1_g3_data + add1_g2_idx * add1_h2_diff * add1_g2_data + add1_g1_idx * add1_h1_diff * add1_g1_data ;

	
	//Hdiff[n*channels*height*width + c*height*width + h*width + w]=0;
	set_data(Hdiff,num,channels,height,width,n,c,h,w,h_diff); 


	//x(t)_diff=(1-g(t))*h(t)_diff
	Dtype g1_idx = get_gate(Idx,num,3,height,width,n,0,h,w,h-1,w-1,horizontal,reverse);
	Dtype g2_idx = get_gate(Idx,num,3,height,width,n,1,h,w,h,w-1,horizontal,reverse);
	Dtype g3_idx = get_gate(Idx,num,3,height,width,n,2,h,w,h+1,w-1,horizontal,reverse);

	Dtype g1_data =  get_gate(G1,num,channels,height,width,n,c,h,w,h-1,w-1,horizontal,reverse);
	Dtype g2_data =  get_gate(G2,num,channels,height,width,n,c,h,w,h,w-1,horizontal,reverse);
	Dtype g3_data =  get_gate(G3,num,channels,height,width,n,c,h,w,h+1,w-1,horizontal,reverse);

	Dtype x_diff = (1-g1_data)*h_diff*g1_idx + (1-g2_data)*h_diff*g2_idx + (1-g3_data)*h_diff*g3_idx;
	set_data(X_diff,num,channels,height,width,n,c,h,w,x_diff);
	

	//g(t)_diff = h(t)_diff * x(t) * -1
	//g(t)_diff+=h(t)_diff * h(t-1)if t>0
	Dtype g1_diff = h_diff * g1_idx * x_data * -1;
	Dtype h1_minus1_data = get_data(H,num,channels,height,width,n,c,h-1,w-1); 
	g1_diff = g1_diff + h_diff * g1_idx*h1_minus1_data;
	set_gate(G1_diff,num,channels,height,width,n,c,h,w,h-1,w-1,horizontal,reverse,g1_diff);

	Dtype g2_diff = h_diff * g2_idx * x_data * -1;
	Dtype h2_minus1_data = get_data(H,num,channels,height,width,n,c,h,w-1); 
	g2_diff = g2_diff + h_diff * g2_idx*h2_minus1_data;
	set_gate(G2_diff,num,channels,height,width,n,c,h,w,h,w-1,horizontal,reverse,g2_diff);

	Dtype g3_diff = h_diff * g3_idx * x_data * -1;
	Dtype h3_minus1_data = get_data(H,num,channels,height,width,n,c,h+1,w-1); 
	g3_diff = g3_diff + h_diff * g3_idx*h3_minus1_data;
	set_gate(G3_diff,num,channels,height,width,n,c,h,w,h+1,w-1,horizontal,reverse,g3_diff);
	
	//idx_diff = h_diff*( (1-g(t))*x(t) + g(t)*h(t-1)  )
	Dtype g1_idx_diff = h_diff * (  (1-g1_data)*x_data + g1_data*h1_minus1_data);
	set_gate_add(Idx_diff,num,3,height,width,n,0,h,w,h-1,w-1,horizontal,reverse,g1_idx_diff);

	Dtype g2_idx_diff = h_diff * (  (1-g2_data)*x_data + g2_data*h2_minus1_data);
	set_gate_add(Idx_diff,num,3,height,width,n,1,h,w,h,w-1,horizontal,reverse,g2_idx_diff);

	Dtype g3_idx_diff = h_diff * (  (1-g3_data)*x_data + g3_data*h3_minus1_data);
	set_gate_add(Idx_diff,num,3,height,width,n,2,h,w,h+1,w-1,horizontal,reverse,g3_idx_diff);
	
	
}
}




template <typename Dtype>
__global__ void backward_one_col_right_left(const int count, int T, int num,int channels, int height,  int width,const Dtype* X,const Dtype* G1, const Dtype* G2,const Dtype* G3,const Dtype* Idx, const Dtype* H, Dtype * X_diff, Dtype * G1_diff,Dtype* G2_diff,Dtype * G3_diff,Dtype * Idx_diff,  Dtype * Hdiff,bool horizontal,bool reverse) {
CUDA_KERNEL_LOOP(index, count) {

	
	int hc_count = height * channels;

	int n,c,h,w;
	int temp=index;



	w = T;
	n = temp / hc_count;
	temp = temp % hc_count;
	c = temp / height;
	temp = temp % height;
	h = temp;

	
	

	Dtype x_data = get_data(X,num,channels,height,width,n,c,h,w);

	
	

	//h(t)_diff = top(t)_diff
	Dtype h_diff = get_data(Hdiff,num,channels,height,width,n,c,h,w); 

	//h(t)_diff += h(t+1)_diff * g(t+1) if t<T
	Dtype add1_g3_idx = get_gate(Idx,num,3,height,width,n,2,h,w,h-1,w-1,horizontal,reverse);
	Dtype add1_h3_diff = get_data(Hdiff,num,channels,height,width,n,c,h-1,w-1);
	Dtype add1_g3_data = get_gate(G3,num,channels,height,width,n,c,h,w,h-1,w-1,horizontal,reverse);

	Dtype add1_g2_idx = get_gate(Idx,num,3,height,width,n,1,h,w,h,w-1,horizontal,reverse);
	Dtype add1_h2_diff = get_data(Hdiff,num,channels,height,width,n,c,h,w-1);
	Dtype add1_g2_data = get_gate(G2,num,channels,height,width,n,c,h,w,h,w-1,horizontal,reverse);

	Dtype add1_g1_idx = get_gate(Idx,num,3,height,width,n,0,h,w,h+1,w-1,horizontal,reverse);
	Dtype add1_h1_diff = get_data(Hdiff,num,channels,height,width,n,c,h+1,w-1);
	Dtype add1_g1_data = get_gate(G1,num,channels,height,width,n,c,h,w,h+1,w-1,horizontal,reverse);

	h_diff = h_diff + add1_g3_idx * add1_h3_diff * add1_g3_data + add1_g2_idx * add1_h2_diff * add1_g2_data + add1_g1_idx * add1_h1_diff * add1_g1_data ;

	
	//Hdiff[n*channels*height*width + c*height*width + h*width + w]=0;
	set_data(Hdiff,num,channels,height,width,n,c,h,w,h_diff); 


	//x(t)_diff=(1-g(t))*h(t)_diff
	Dtype g1_idx = get_gate(Idx,num,3,height,width,n,0,h,w,h-1,w+1,horizontal,reverse);
	Dtype g2_idx = get_gate(Idx,num,3,height,width,n,1,h,w,h,w+1,horizontal,reverse);
	Dtype g3_idx = get_gate(Idx,num,3,height,width,n,2,h,w,h+1,w+1,horizontal,reverse);

	Dtype g1_data =  get_gate(G1,num,channels,height,width,n,c,h,w,h-1,w+1,horizontal,reverse);
	Dtype g2_data =  get_gate(G2,num,channels,height,width,n,c,h,w,h,w+1,horizontal,reverse);
	Dtype g3_data =  get_gate(G3,num,channels,height,width,n,c,h,w,h+1,w+1,horizontal,reverse);

	Dtype x_diff = (1-g1_data)*h_diff*g1_idx + (1-g2_data)*h_diff*g2_idx + (1-g3_data)*h_diff*g3_idx;
	set_data(X_diff,num,channels,height,width,n,c,h,w,x_diff);
	

	//g(t)_diff = h(t)_diff * x(t) * -1
	//g(t)_diff+=h(t)_diff * h(t-1)if t>0
	Dtype g1_diff = h_diff * g1_idx * x_data * -1;
	Dtype h1_minus1_data = get_data(H,num,channels,height,width,n,c,h-1,w+1); 
	g1_diff = g1_diff + h_diff * g1_idx*h1_minus1_data;
	set_gate(G1_diff,num,channels,height,width,n,c,h,w,h-1,w+1,horizontal,reverse,g1_diff);

	Dtype g2_diff = h_diff * g2_idx * x_data * -1;
	Dtype h2_minus1_data = get_data(H,num,channels,height,width,n,c,h,w+1); 
	g2_diff = g2_diff + h_diff * g2_idx*h2_minus1_data;
	set_gate(G2_diff,num,channels,height,width,n,c,h,w,h,w+1,horizontal,reverse,g2_diff);

	Dtype g3_diff = h_diff * g3_idx * x_data * -1;
	Dtype h3_minus1_data = get_data(H,num,channels,height,width,n,c,h+1,w+1); 
	g3_diff = g3_diff + h_diff * g3_idx*h3_minus1_data;
	set_gate(G3_diff,num,channels,height,width,n,c,h,w,h+1,w+1,horizontal,reverse,g3_diff);
	
	//idx_diff = h_diff*( (1-g(t))*x(t) + g(t)*h(t-1)  )
	Dtype g1_idx_diff = h_diff * (  (1-g1_data)*x_data + g1_data*h1_minus1_data);
	set_gate_add(Idx_diff,num,3,height,width,n,0,h,w,h-1,w+1,horizontal,reverse,g1_idx_diff);

	Dtype g2_idx_diff = h_diff * (  (1-g2_data)*x_data + g2_data*h2_minus1_data);
	set_gate_add(Idx_diff,num,3,height,width,n,1,h,w,h,w+1,horizontal,reverse,g2_idx_diff);

	Dtype g3_idx_diff = h_diff * (  (1-g3_data)*x_data + g3_data*h3_minus1_data);
	set_gate_add(Idx_diff,num,3,height,width,n,2,h,w,h+1,w+1,horizontal,reverse,g3_idx_diff);
	
	
}
}



template <typename Dtype>
__global__ void backward_one_row_top_bottom(const int count, int T, int num,int channels, int height,  int width,const Dtype* X,const Dtype* G1, const Dtype* G2,const Dtype* G3,const Dtype* Idx, const Dtype* H, Dtype * X_diff, Dtype * G1_diff,Dtype* G2_diff,Dtype * G3_diff,Dtype * Idx_diff,  Dtype * Hdiff,bool horizontal,bool reverse) {
CUDA_KERNEL_LOOP(index, count) {

	
	int wc_count = width * channels;

	int n,c,h,w;
	int temp=index;
	h = T;
	n = temp / wc_count;
	temp = temp % wc_count;
	c = temp / width;
	temp = temp % width;
	w = temp;

	Dtype x_data = get_data(X,num,channels,height,width,n,c,h,w);

	//h(t)_diff = top(t)_diff
	Dtype h_diff = get_data(Hdiff,num,channels,height,width,n,c,h,w); 

	//h(t)_diff += h(t+1)_diff * g(t+1) if t<T
	Dtype add1_g3_idx = get_gate(Idx,num,3,height,width,n,2,h,w,h+1,w-1,horizontal,reverse);
	Dtype add1_h3_diff = get_data(Hdiff,num,channels,height,width,n,c,h+1,w-1);
	Dtype add1_g3_data = get_gate(G3,num,channels,height,width,n,c,h,w,h+1,w-1,horizontal,reverse);

	Dtype add1_g2_idx = get_gate(Idx,num,3,height,width,n,1,h,w,h+1,w,horizontal,reverse);
	Dtype add1_h2_diff = get_data(Hdiff,num,channels,height,width,n,c,h+1,w);
	Dtype add1_g2_data = get_gate(G2,num,channels,height,width,n,c,h,w,h+1,w,horizontal,reverse);

	Dtype add1_g1_idx = get_gate(Idx,num,3,height,width,n,0,h,w,h+1,w+1,horizontal,reverse);
	Dtype add1_h1_diff = get_data(Hdiff,num,channels,height,width,n,c,h+1,w+1);
	Dtype add1_g1_data = get_gate(G1,num,channels,height,width,n,c,h,w,h+1,w+1,horizontal,reverse);

	h_diff = h_diff + add1_g3_idx * add1_h3_diff * add1_g3_data + add1_g2_idx * add1_h2_diff * add1_g2_data + add1_g1_idx * add1_h1_diff * add1_g1_data ;

	
	//Hdiff[n*channels*height*width + c*height*width + h*width + w]=0;
	set_data(Hdiff,num,channels,height,width,n,c,h,w,h_diff); 


	//x(t)_diff=(1-g(t))*h(t)_diff
	Dtype g1_idx = get_gate(Idx,num,3,height,width,n,0,h,w,h-1,w-1,horizontal,reverse);
	Dtype g2_idx = get_gate(Idx,num,3,height,width,n,1,h,w,h-1,w,horizontal,reverse);
	Dtype g3_idx = get_gate(Idx,num,3,height,width,n,2,h,w,h-1,w+1,horizontal,reverse);

	Dtype g1_data =  get_gate(G1,num,channels,height,width,n,c,h,w,h-1,w-1,horizontal,reverse);
	Dtype g2_data =  get_gate(G2,num,channels,height,width,n,c,h,w,h-1,w,horizontal,reverse);
	Dtype g3_data =  get_gate(G3,num,channels,height,width,n,c,h,w,h-1,w+1,horizontal,reverse);

	Dtype x_diff = (1-g1_data)*h_diff*g1_idx + (1-g2_data)*h_diff*g2_idx + (1-g3_data)*h_diff*g3_idx;
	set_data(X_diff,num,channels,height,width,n,c,h,w,x_diff);
	

	//g(t)_diff = h(t)_diff * x(t) * -1
	//g(t)_diff+=h(t)_diff * h(t-1)if t>0
	Dtype g1_diff = h_diff * g1_idx * x_data * -1;
	Dtype h1_minus1_data = get_data(H,num,channels,height,width,n,c,h-1,w-1); 
	g1_diff = g1_diff + h_diff * g1_idx*h1_minus1_data;
	set_gate(G1_diff,num,channels,height,width,n,c,h,w,h-1,w-1,horizontal,reverse,g1_diff);

	Dtype g2_diff = h_diff * g2_idx * x_data * -1;
	Dtype h2_minus1_data = get_data(H,num,channels,height,width,n,c,h-1,w); 
	g2_diff = g2_diff + h_diff * g2_idx*h2_minus1_data;
	set_gate(G2_diff,num,channels,height,width,n,c,h,w,h-1,w,horizontal,reverse,g2_diff);

	Dtype g3_diff = h_diff * g3_idx * x_data * -1;
	Dtype h3_minus1_data = get_data(H,num,channels,height,width,n,c,h-1,w+1); 
	g3_diff = g3_diff + h_diff * g3_idx*h3_minus1_data;
	set_gate(G3_diff,num,channels,height,width,n,c,h,w,h-1,w+1,horizontal,reverse,g3_diff);
	
	//idx_diff = h_diff*( (1-g(t))*x(t) + g(t)*h(t-1)  )
	Dtype g1_idx_diff = h_diff * (  (1-g1_data)*x_data + g1_data*h1_minus1_data);
	set_gate_add(Idx_diff,num,3,height,width,n,0,h,w,h-1,w-1,horizontal,reverse,g1_idx_diff);

	Dtype g2_idx_diff = h_diff * (  (1-g2_data)*x_data + g2_data*h2_minus1_data);
	set_gate_add(Idx_diff,num,3,height,width,n,1,h,w,h-1,w,horizontal,reverse,g2_idx_diff);

	Dtype g3_idx_diff = h_diff * (  (1-g3_data)*x_data + g3_data*h3_minus1_data);
	set_gate_add(Idx_diff,num,3,height,width,n,2,h,w,h-1,w+1,horizontal,reverse,g3_idx_diff);
	
	
}
}



template <typename Dtype>
__global__ void backward_one_row_bottom_top(const int count, int T, int num,int channels, int height,  int width,const Dtype* X,const Dtype* G1, const Dtype* G2,const Dtype* G3,const Dtype* Idx, const Dtype* H, Dtype * X_diff, Dtype * G1_diff,Dtype* G2_diff,Dtype * G3_diff,Dtype * Idx_diff,  Dtype * Hdiff,bool horizontal,bool reverse) {
CUDA_KERNEL_LOOP(index, count) {

	
	int wc_count = width * channels;

	int n,c,h,w;
	int temp=index;
	h = T;
	n = temp / wc_count;
	temp = temp % wc_count;
	c = temp / width;
	temp = temp % width;
	w = temp;

	Dtype x_data = get_data(X,num,channels,height,width,n,c,h,w);

	//h(t)_diff = top(t)_diff
	Dtype h_diff = get_data(Hdiff,num,channels,height,width,n,c,h,w); 

	//h(t)_diff += h(t+1)_diff * g(t+1) if t<T
	Dtype add1_g3_idx = get_gate(Idx,num,3,height,width,n,2,h,w,h-1,w-1,horizontal,reverse);
	Dtype add1_h3_diff = get_data(Hdiff,num,channels,height,width,n,c,h-1,w-1);
	Dtype add1_g3_data = get_gate(G3,num,channels,height,width,n,c,h,w,h-1,w-1,horizontal,reverse);

	Dtype add1_g2_idx = get_gate(Idx,num,3,height,width,n,1,h,w,h-1,w,horizontal,reverse);
	Dtype add1_h2_diff = get_data(Hdiff,num,channels,height,width,n,c,h-1,w);
	Dtype add1_g2_data = get_gate(G2,num,channels,height,width,n,c,h,w,h-1,w,horizontal,reverse);

	Dtype add1_g1_idx = get_gate(Idx,num,3,height,width,n,0,h,w,h-1,w+1,horizontal,reverse);
	Dtype add1_h1_diff = get_data(Hdiff,num,channels,height,width,n,c,h-1,w+1);
	Dtype add1_g1_data = get_gate(G1,num,channels,height,width,n,c,h,w,h-1,w+1,horizontal,reverse);

	h_diff = h_diff + add1_g3_idx * add1_h3_diff * add1_g3_data + add1_g2_idx * add1_h2_diff * add1_g2_data + add1_g1_idx * add1_h1_diff * add1_g1_data ;

	
	//Hdiff[n*channels*height*width + c*height*width + h*width + w]=0;
	set_data(Hdiff,num,channels,height,width,n,c,h,w,h_diff); 


	//x(t)_diff=(1-g(t))*h(t)_diff
	Dtype g1_idx = get_gate(Idx,num,3,height,width,n,0,h,w,h+1,w-1,horizontal,reverse);
	Dtype g2_idx = get_gate(Idx,num,3,height,width,n,1,h,w,h+1,w,horizontal,reverse);
	Dtype g3_idx = get_gate(Idx,num,3,height,width,n,2,h,w,h+1,w+1,horizontal,reverse);

	Dtype g1_data =  get_gate(G1,num,channels,height,width,n,c,h,w,h+1,w-1,horizontal,reverse);
	Dtype g2_data =  get_gate(G2,num,channels,height,width,n,c,h,w,h+1,w,horizontal,reverse);
	Dtype g3_data =  get_gate(G3,num,channels,height,width,n,c,h,w,h+1,w+1,horizontal,reverse);

	Dtype x_diff = (1-g1_data)*h_diff*g1_idx + (1-g2_data)*h_diff*g2_idx + (1-g3_data)*h_diff*g3_idx;
	set_data(X_diff,num,channels,height,width,n,c,h,w,x_diff);
	

	//g(t)_diff = h(t)_diff * x(t) * -1
	//g(t)_diff+=h(t)_diff * h(t-1)if t>0
	Dtype g1_diff = h_diff * g1_idx * x_data * -1;
	Dtype h1_minus1_data = get_data(H,num,channels,height,width,n,c,h+1,w-1); 
	g1_diff = g1_diff + h_diff * g1_idx*h1_minus1_data;
	set_gate(G1_diff,num,channels,height,width,n,c,h,w,h+1,w-1,horizontal,reverse,g1_diff);

	Dtype g2_diff = h_diff * g2_idx * x_data * -1;
	Dtype h2_minus1_data = get_data(H,num,channels,height,width,n,c,h+1,w); 
	g2_diff = g2_diff + h_diff * g2_idx*h2_minus1_data;
	set_gate(G2_diff,num,channels,height,width,n,c,h,w,h+1,w,horizontal,reverse,g2_diff);

	Dtype g3_diff = h_diff * g3_idx * x_data * -1;
	Dtype h3_minus1_data = get_data(H,num,channels,height,width,n,c,h+1,w+1); 
	g3_diff = g3_diff + h_diff * g3_idx*h3_minus1_data;
	set_gate(G3_diff,num,channels,height,width,n,c,h,w,h+1,w+1,horizontal,reverse,g3_diff);
	
	//idx_diff = h_diff*( (1-g(t))*x(t) + g(t)*h(t-1)  )
	Dtype g1_idx_diff = h_diff * (  (1-g1_data)*x_data + g1_data*h1_minus1_data);
	set_gate_add(Idx_diff,num,3,height,width,n,0,h,w,h+1,w-1,horizontal,reverse,g1_idx_diff);

	Dtype g2_idx_diff = h_diff * (  (1-g2_data)*x_data + g2_data*h2_minus1_data);
	set_gate_add(Idx_diff,num,3,height,width,n,1,h,w,h+1,w,horizontal,reverse,g2_idx_diff);

	Dtype g3_idx_diff = h_diff * (  (1-g3_data)*x_data + g3_data*h3_minus1_data);
	set_gate_add(Idx_diff,num,3,height,width,n,2,h,w,h+1,w+1,horizontal,reverse,g3_idx_diff);
	
	
}
}

template <typename Dtype>
void GateRecurrent2dLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
    
	const Dtype* X = bottom[0]->gpu_data();
	const Dtype* G1 = bottom[1]->gpu_data();
	const Dtype* G2 = bottom[2]->gpu_data();
	const Dtype* G3 = bottom[3]->gpu_data();
	const Dtype* Idx = bottom[4]->gpu_data();
	Dtype * H = top[0]->mutable_gpu_data();

	if(horizontal_ && !reverse_) // left to right
	{
		const int count = height_ * channels_ * num_;

		for(int t=0;t<width_;t++)
		{
			
	
			forward_one_col_left_right<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,t, num_,channels_,height_,width_,X,G1,G2,G3,Idx,H,horizontal_,reverse_);

			CUDA_POST_KERNEL_CHECK;
		}
	}
	else if(horizontal_ && reverse_) // right to left
	{
		const int count = height_ * channels_ * num_;

		for(int t=width_ - 1; t>=0; t--)
		{
			
	
			forward_one_col_right_left<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,t, num_,channels_,height_,width_,X,G1,G2,G3,Idx,H,horizontal_,reverse_);
			CUDA_POST_KERNEL_CHECK;
		}
	}
	else if(!horizontal_ && !reverse_) // top to bottom
	{
		const int count = width_ * channels_ * num_;

		for(int t=0; t< height_; t++)
		{
			
	
			forward_one_row_top_bottom<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,t, num_,channels_,height_,width_,X,G1,G2,G3,Idx,H,horizontal_,reverse_);
			CUDA_POST_KERNEL_CHECK;
		}
	}
	else  //bottom to top
	{
		const int count = width_ * channels_ * num_;

		for(int t=height_-1; t>=0; t--)
		{
			
	
			forward_one_row_bottom_top<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,t, num_,channels_,height_,width_,X,G1,G2,G3,Idx,H,horizontal_,reverse_);
			CUDA_POST_KERNEL_CHECK;
		}
	}


}

template <typename Dtype>
void GateRecurrent2dLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {

    	const Dtype* X = bottom[0]->gpu_data();
	const Dtype* G1 = bottom[1]->gpu_data();
	const Dtype* G2 = bottom[2]->gpu_data();
	const Dtype* G3 = bottom[3]->gpu_data();
	const Dtype* Idx = bottom[4]->gpu_data();
	const Dtype * H = top[0]->gpu_data();

	Dtype * H_diff = H_.mutable_gpu_diff();
	caffe_copy(top[0]->count(),top[0]->gpu_diff(),H_diff);

	Dtype * X_diff = bottom[0]->mutable_gpu_diff();
	Dtype * G1_diff = bottom[1]->mutable_gpu_diff();
	Dtype * G2_diff = bottom[2]->mutable_gpu_diff();
	Dtype * G3_diff = bottom[3]->mutable_gpu_diff();
	Dtype * Idx_diff = bottom[4]->mutable_gpu_diff();

	Dtype * H_cpudiff = H_.mutable_cpu_diff();

	SaveArray("topdiff.txt", H_.mutable_cpu_diff(),top[0]->count());

	if(horizontal_ && ! reverse_) //left to right
	{
		const int count =  height_ * channels_ * num_;

		for(int t = width_ -1; t>=0; t--)
		{
			backward_one_col_left_right<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,t, num_,channels_,height_,width_,X,G1,G2,G3,Idx,H,X_diff,G1_diff,G2_diff,G3_diff,Idx_diff,H_diff,horizontal_, reverse_);

			CUDA_POST_KERNEL_CHECK;

		}
	}
	else if(horizontal_ &&  reverse_) //right to left
	{
		const int count =  height_ * channels_ * num_;

		for(int t = 0; t<width_; t++)
		{
			backward_one_col_right_left<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,t, num_,channels_,height_,width_,X,G1,G2,G3,Idx,H,X_diff,G1_diff,G2_diff,G3_diff,Idx_diff,H_diff,horizontal_, reverse_);

			CUDA_POST_KERNEL_CHECK;

		}
	}
	else if(!horizontal_ &&  !reverse_) //top to bottom
	{
		const int count =  width_ * channels_ * num_;
		for(int t = height_-1; t>=0; t--)
		{
			backward_one_row_top_bottom<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,t, num_,channels_,height_,width_,X,G1,G2,G3,Idx,H,X_diff,G1_diff,G2_diff,G3_diff,Idx_diff,H_diff,horizontal_, reverse_);

			CUDA_POST_KERNEL_CHECK;

		}
	}
	else  //bottom to top
	{
		const int count =  width_ * channels_ * num_;
		for(int t = 0; t<height_; t++)
		{
			backward_one_row_bottom_top<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,t, num_,channels_,height_,width_,X,G1,G2,G3,Idx,H,X_diff,G1_diff,G2_diff,G3_diff,Idx_diff,H_diff,horizontal_, reverse_);

			CUDA_POST_KERNEL_CHECK;

		}
	}

}

INSTANTIATE_LAYER_GPU_FUNCS(GateRecurrent2dLayer);

}  // namespace caffe
