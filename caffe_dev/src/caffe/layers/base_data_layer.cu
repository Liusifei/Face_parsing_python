#include <vector>

#include "caffe/layers/base_data_layer.hpp"

namespace caffe {

template <typename Dtype>
void BasePrefetchingDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  Batch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");
  // Reshape to loaded data.
  top[0]->ReshapeLike(batch->data_);
  // Copy the data
  caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
      top[0]->mutable_gpu_data());
  if (this->output_labels_) {
    // Reshape to loaded labels.
    top[1]->ReshapeLike(batch->label_);
    // Copy the labels.
    caffe_copy(batch->label_.count(), batch->label_.gpu_data(),
        top[1]->mutable_gpu_data());
  }
  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
  prefetch_free_.push(batch);
}


template <typename Dtype>
void BasePrefetchingArbitraryDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  LiangjiBatch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");
  for(int i=0;i<batch->blobs_.size();i++)
  {
      CHECK(top[i]->ShapeEquals(batch->blobs_[i]));
      //top[i]->ReshapeLike(batch->blobs_[i]);
      caffe_copy(batch->blobs_[i]->count(), batch->blobs_[i]->gpu_data(),
             top[i]->mutable_gpu_data());
  }
  
  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
  prefetch_free_.push(batch);
}

INSTANTIATE_LAYER_GPU_FORWARD(BasePrefetchingDataLayer);
INSTANTIATE_LAYER_GPU_FORWARD(BasePrefetchingArbitraryDataLayer);

}  // namespace caffe
