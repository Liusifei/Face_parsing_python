/*
 * Author: Liangji 
 * Email: liangji20040249@gmail.com
*/
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/manipulateloss_layer.hpp"

namespace caffe {

template <typename Dtype>
void ManipulateLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
    
    caffe_copy(bottom[0]->count(), bottom[0]->gpu_data(), top[0]->mutable_gpu_data());
}

template <typename Dtype>
void ManipulateLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    
    Backward_cpu( top, propagate_down, bottom) ;
}

INSTANTIATE_LAYER_GPU_FUNCS(ManipulateLossLayer);

}  // namespace caffe
