#include "hip/hip_runtime.h"
/*
File name: weightwrap_layer.cpp
by Guangyu Zhong
(guangyuzhonghikari@gmail.com)
Date: 02/06/2018

Example:
Input feat: N*32*H*W feature map
      left: -1
      top: -1
      output: wrap(feat to 1, 1) dot feat, means each node dot the left top one, i.e., the distance btw each node and its left top node.
      N * 32 * H * W
*/
#include <vector>
//#include "caffe/blob.hpp"
//#include "caffe/common.hpp"
//#include "caffe/filler.hpp"
//#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
//#include "caffe/util/thread_functions.hpp"
#include "caffe/layers/weightwrap_layer.hpp"
//#include "caffe/util/io.hpp"

namespace caffe {

    #define min(a,b) ((a<b)?(a):(b))
    #define max(a,b) ((a>b)?(a):(b))
    // sifeiliu: add a flag
    __device__ int get_translate_index(int index, int channels,int height, int width, int horizontal, int vertical){
        int n = index / (channels * height * width);
        int c = (index - n * (channels * height * width)) / (height * width);
        int h = (index - n * (channels * height * width) - c * (height * width)) / width;
        int w = index - n * (channels * height * width) - c * (height * width) - h * width;

        w = w + horizontal;
        h = h + vertical;

        if ((w >= 0 && w < width) && (h >=0 && h < height))
        	return (n * channels * height * width + c * height * width + h * width + w); //((n * channels + c) * height + h) * width + w.
        else
            return -1;
    }



    template <typename Dtype>
    __global__ void zero_blob(Dtype* input, const int count){
        CUDA_KERNEL_LOOP(index, count){
            input[index] = 0;
        }
    }

    /* sifeiliu: 
        1. we need to avoid duplicated boundaries so that thoes pixels can copy from the prior. 2. change output to num * height * width.
        3. output is inited as zeros.
    */
    template <typename Dtype>
    __global__ void forward_wrap_dot_matrix(const Dtype* input, Dtype* output, int count, int channels,int height, int width, int horizontal, int vertical){
        CUDA_KERNEL_LOOP(index, count){
            int new_index = get_translate_index(index, channels, height, width, horizontal, vertical);
            // int index_reduce = get_translate_channel(index, num, channels, height, width);
            if (new_index!=-1)
                output[index] = input[new_index] * input[index];
        }
    }


    /* sifeiliu: 
        1. revise as the same;
        2. X_diff is inited as zeros;
        //(const Dtype *, const Dtype *, int, int, int, int, int, int) [with Dtype=float]
    */
    template <typename Dtype>
    __global__ void backward_wrap_dot_matrix(const Dtype* W_diff, Dtype* X1_diff, Dtype* X2_diff, const Dtype* X, int count, int channels,int height, int width, int horizontal, int vertical){
        
        //int num = count / (channels * height * width);

        CUDA_KERNEL_LOOP(index, count){
            int new_index = get_translate_index(index, channels, height, width, horizontal, vertical);
            if (new_index!=-1) {    
                // int index_reduce = get_translate_channel(index, num, channels, height, width);
                X1_diff[index] = X[new_index] * W_diff[index];
                X2_diff[new_index] = X[index] * W_diff[index];
            }
        }
    }


    /* sifeiliu:
        1. revise the size of trans
        2. trans init to zero
    */
    template <typename Dtype>
    void WeightWrapLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
        const int count = bottom[0]->count();
    	const Dtype* X = bottom[0]->gpu_data();

    	Dtype* trans = top[0]->mutable_gpu_data();
        zero_blob<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(trans, count);
        CUDA_POST_KERNEL_CHECK;

        const int width = bottom[0]->width();
        const int height = bottom[0]->height();
        const int channels = bottom[0]->channels();
        const int num = bottom[0]->num();
        const int horizontal = horizontal_;
        const int vertical = vertical_;
        forward_wrap_dot_matrix<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(X, trans, count, channels, height, width, horizontal, vertical);
        CUDA_POST_KERNEL_CHECK;
    }

    
    template <typename Dtype>
    void WeightWrapLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
        const int count = bottom[0]->count();
    	const Dtype* X = bottom[0]->gpu_data(); // source feature map
        const Dtype* W = top[0]->gpu_data();
        const Dtype* W_diff = top[0]->gpu_diff();
        Dtype* X_diff = bottom[0]->mutable_gpu_diff();
        const int width = bottom[0]->width();
        const int height = bottom[0]->height();
        const int channels = bottom[0]->channels();
        const int num = bottom[0]->num();
        const int horizontal = horizontal_;
        const int vertical = vertical_;

        Blob<Dtype> x1_diff(num, channels, height, width);
        Blob<Dtype> x2_diff(num, channels, height, width);
        Dtype* X1_diff = x1_diff.mutable_gpu_diff();
        Dtype* X2_diff = x2_diff.mutable_gpu_diff();

        backward_wrap_dot_matrix<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(W_diff, X1_diff, X2_diff, X, count, channels, height, width, horizontal, vertical);
        CUDA_POST_KERNEL_CHECK;
        caffe_gpu_add(count, x1_diff.gpu_diff(), x2_diff.gpu_diff(), X1_diff);
    }

INSTANTIATE_LAYER_GPU_FUNCS(WeightWrapLayer);
}
